#include "myHeaders.hpp"

// int PRINT_DEBUG = 0;

int
main(int argc, char** argv)
{
  // Host problem definition
  unsigned int A_rows = atoi(argv[1]);
  unsigned int A_cols = atoi(argv[2]);
  float threshold = atof(argv[3]);
  int PRINT_DEBUG = atoi(argv[4]);

  unsigned int B_rows = A_cols;
  unsigned int B_cols = A_cols;
  unsigned int C_rows = A_rows;
  unsigned int C_cols = B_cols;
  unsigned int lda = A_cols;
  unsigned int ldb = B_cols;
  unsigned int ldc = C_cols;

  torch::Tensor A = torch::randn({A_rows, A_cols}, torch::dtype(torch::kFloat32));
  torch::Tensor B = torch::randn({B_rows, B_cols}, torch::dtype(torch::kFloat32));
  A.masked_fill_(A < threshold, 0);
  torch::Tensor C = torch::zeros({A_rows, B_cols});

  float alpha = 1.0f;
  float beta  = 0.0f;


  float *hA = A.contiguous().data_ptr<float>();
  float *hB = B.contiguous().data_ptr<float>();
  float *hC = C.contiguous().data_ptr<float>();

  // count how many non zero values A has
  unsigned int n_non_zeroes = 0;
  count_non_zeroes(hA, A_rows, A_cols, &n_non_zeroes);
  printf("number of non zeroes in A: %d\n", n_non_zeroes);

  // put the non zero values of A into a contiguous array
  float *non_zero_values = (float*) malloc(n_non_zeroes*sizeof(float));
  extract_non_zeros(hA, A_rows, A_cols, non_zero_values);

  // Get the ellColInd array for matrix A
  int ellBlockSize, ellCols, err;
  int* ellColInd = nullptr;
  float* ellValue = nullptr; // We don't care about this at the moment

  err = getBellParams(A, A_rows, A_cols, ellBlockSize, ellCols, ellColInd, ellValue);
  if (err != 0)
  {
    printf("Error code %d, exiting!\n", err);
    fflush(stdout);
    return err;
  }

  // ATTENTION: ellCols is usually considered to be the number of columns in ell format, NOT the number of blocks.
  ellCols = ellBlockSize * ellCols;
  // Device memory management
  // printf("ellCols: %d, n_non_zeroes: %d\n", ellCols, n_non_zeroes);
  int ellColInd_size = A_rows * ellCols;
  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream))

  int *dA_columns;
  float *dA_values, *dB, *dC;
  CHECK_CUDA(hipMallocAsync((void**) &dA_columns, ellColInd_size * sizeof(int), stream))
  CHECK_CUDA(hipMallocAsync((void**) &dA_values, A_rows * ellCols * sizeof(float), stream))
  CHECK_CUDA(hipMallocAsync((void**) &dB, B_rows * B_cols * sizeof(float), stream))
  CHECK_CUDA(hipMallocAsync((void**) &dC, C_rows * C_cols * sizeof(float), stream))
  CHECK_CUDA(hipMemcpyAsync(dA_columns, ellColInd, ellColInd_size * sizeof(int), hipMemcpyHostToDevice, stream))
  // CHECK_CUDA(hipMemcpy(dA_values, ellValue, A_rows * ellCols * sizeof(float), hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemsetAsync(dA_values, 0.0f, A_rows * ellCols * sizeof(float), stream))
  CHECK_CUDA(hipMemcpyAsync(dB, hB, B_rows * B_cols * sizeof(float), hipMemcpyHostToDevice, stream))
  CHECK_CUDA(hipMemcpyAsync(dC, hC, C_rows * C_cols * sizeof(float), hipMemcpyHostToDevice, stream))
  CHECK_CUDA(hipStreamSynchronize(stream))

  printf("ellCols: %d, ellBlockSize: %d\n", ellCols, ellBlockSize);

  /* [BEGIN] Dense to sparse conversion */
  // To create a conversion you need a dense matrix to convert it into a sparse matrix. If you want to store matrix A
  // in a sparse format, you need to convert A's dense representation to sparse!
  hipsparseHandle_t     conversionHandle = NULL;
  hipsparseDnMatDescr_t matA;
  hipsparseSpMatDescr_t matSpA;
  void*                dBuffer    = NULL;
  size_t               bufferSize = 0;
  CHECK_CUSPARSE(hipsparseCreate(&conversionHandle))

  /* [BEGIN] Create events to time the runtime of spmm */
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start))
  CHECK_CUDA(hipEventCreate(&stop))
  /* [END] Create events to time the runtime of spmm */



  /* ATTENTION: remember that leading dimension is number of columns if we use HIPSPARSE_ORDER_ROW, and vice versa */
  // Create dense matrix A
  float *dA_dense;
  CHECK_CUDA(hipMallocAsync((void**) &dA_dense, A_rows * A_cols * sizeof(double), stream))
  CHECK_CUDA(hipMemcpyAsync(dA_dense, hA, A_rows * A_cols * sizeof(double), hipMemcpyHostToDevice, stream))
  CHECK_CUDA(hipStreamSynchronize(stream))

  CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_rows, A_cols, lda, dA_dense,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )

  // Create sparse matrix B in Blocked ELL format
  CHECK_CUSPARSE( hipsparseCreateBlockedEll(&matSpA, A_rows, A_cols,
                                           ellBlockSize, ellCols,
                                           dA_columns, dA_values,
                                           HIPSPARSE_INDEX_32I,
                                           HIPSPARSE_INDEX_BASE_ZERO,
                                           HIP_R_32F) )

  // allocate an external buffer if needed
  CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(conversionHandle, matA, matSpA,
                                                  HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

  // execute Sparse to Dense conversion
  CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(conversionHandle, matA, matSpA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer))

  // execute Sparse to Dense conversion
  CHECK_CUSPARSE(hipsparseDenseToSparse_convert(conversionHandle, matA, matSpA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer))
  /* [END] Dense to sparse conversion */


  /* [BEGIN] Execute sparse-dense matrix multiplication */

  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, B_rows, B_cols, ldb, dB,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )
  CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, C_rows, C_cols, ldc, dC,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW) )

  execute_spmm<float>(matSpA, matB, matC, alpha, beta);


  /* [END] Execute sparse-dense matrix multiplication */

  float *h_ellValues = (float*) malloc(A_rows * ellCols * sizeof(float));
  CHECK_CUDA(hipMemcpy(h_ellValues, dA_values, A_rows * ellCols * sizeof(float), hipMemcpyDeviceToHost))
  for (unsigned int i = 0; i < A_rows; ++i)
  {
    for (unsigned int j = 0; j < ellCols; ++j)
    {
      printf("%f ", h_ellValues[i * ellCols + j]);
    }
    printf("\n");
  }
  if (PRINT_DEBUG > 0)
  {
    CHECK_CUDA(hipMemcpy(hC, dC, C_rows * C_cols * sizeof(float), hipMemcpyDeviceToHost))
    printf("SpMM result:\n");
    for (unsigned int i = 0; i < C_rows; ++i)
    {
      for (unsigned int j = 0; j < C_cols; ++j)
      {
        printf("%f ", hC[i * C_cols + j]);
      }
      printf("\n");
    }

    torch::Tensor res = torch::mm(A,B);
    printf("PyTorch result:\n");
    std::cout << res << std::endl;
  }


  CHECK_CUDA(hipFree(dA_columns))
  CHECK_CUDA(hipFree(dA_values))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(dC))
  CHECK_CUDA(hipFree(dBuffer))
  CHECK_CUDA(hipFree(dA_dense))
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matSpA))
  CHECK_CUSPARSE(hipsparseDestroy(conversionHandle))
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
  CHECK_CUDA(hipEventDestroy(start))
  CHECK_CUDA(hipEventDestroy(stop))
  free(non_zero_values);
  free(ellColInd);
  free(ellValue);
  free(h_ellValues);
  return EXIT_SUCCESS;
}


